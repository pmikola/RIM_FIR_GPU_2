#include "hip/hip_runtime.h"
﻿// RIM_FIR_CPU.cpp : This file contains the 'main' function. Program execution begins and ends there.
//
#define K 512

#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>


__global__ static void audiofir_kernel(
	float* yout, float* yin, float* coeff, int n, int len)
{
	float output = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		for (int k = 0; k <= n; k++) {
			if (i >= k) output += yin[i - k] * coeff[k];
		}
		yout[i] = output;
	}
}


void audiofir(float* yout, float* yin, float* coeff, int n, int len, ...)
{

	float* coeffd, * yind, * youtd;
	auto M = (len + K - 1) / K;
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipMalloc(&coeffd, sizeof(float) * (n + 1)));
	checkCudaErrors(hipMalloc(&yind, sizeof(float) * (2 * len)));
	checkCudaErrors(hipMalloc(&youtd, sizeof(float) * (2 * len)));
	checkCudaErrors(hipMemcpy(coeffd, coeff, sizeof(float) * (n + 1), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(yind, yin, sizeof(float) * (2 * len), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy(&youtd, yout, 2 * len, hipMemcpyHostToDevice));
	hipEvent_t start1, stop1; // pomiar czasu wykonania jądra
	checkCudaErrors(hipEventCreate(&start1));
	checkCudaErrors(hipEventCreate(&stop1));
	checkCudaErrors(hipEventRecord(start1, 0));
	audiofir_kernel << <M, K >> > (youtd, yind, coeffd, n, len);
	audiofir_kernel << <M, K >> > (youtd + len, yind + len, coeffd, n, len);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventRecord(stop1, 0));
	checkCudaErrors(hipEventSynchronize(stop1));
	float elapsedTime;
	checkCudaErrors(hipEventElapsedTime(&elapsedTime,
		start1, stop1));
	checkCudaErrors(hipEventDestroy(start1));
	checkCudaErrors(hipEventDestroy(stop1));
	checkCudaErrors(hipDeviceSynchronize());
	printf("GPU (kernel) time = %.3f ms (%6.3f GFLOP/s)\n",
		elapsedTime,
		1e-6 * 2 * ((double)n + 1) * 2 * ((double)len) /
		elapsedTime);
	checkCudaErrors(hipMemcpy(&yout, youtd, 2 * len, hipMemcpyDeviceToHost));
	if (IsDebuggerPresent()) getchar();
	hipFree(&coeffd);
	hipFree(&yind);
	hipFree(&youtd);
}